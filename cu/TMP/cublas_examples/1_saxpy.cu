#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define N 1000000

#define checkCUDA(expression)                  \
{                                              \
	hipError_t status = (expression);           \
	if (status != hipSuccess) {                 \
		printf("Error on line %d: err code %d\n",  \
				__LINE__, status);                     \
		exit(EXIT_FAILURE);                        \
	}                                            \
}

#define checkCUBLAS(expression)                \
{                                              \
	hipblasStatus_t status = (expression);        \
	if (status != HIPBLAS_STATUS_SUCCESS) {       \
		printf("Error on line %d: err code %d\n",  \
				__LINE__, status);                     \
		exit(EXIT_FAILURE);                        \
	}                                            \
}

#define getMillisecond(start, end) \
	(end.tv_sec-start.tv_sec)*1000 + \
	(end.tv_usec-start.tv_usec)/1000.0


int main (void){
	hipblasHandle_t handle;
	float *x, *y, *resultCPU, *resultGPU;
	float *devPtrX, *devPtrY;
	float alpha = 1.2;
	float incx = 1, incy = 1;

	float ms = 0;
	struct timeval start, end;
	srand(2018);

	// Memory for host
	x = (float *)malloc (N * sizeof (float));
	y = (float *)malloc (N * sizeof (float));
	resultCPU = (float *)malloc (N * sizeof (float));
	resultGPU = (float *)malloc (N * sizeof (float));

	// Init values
	for (int i = 0; i < N; i++) {
		x[i] = (rand() % 1000000) / 10000.0;
		y[i] = (rand() % 1000000) / 10000.0;
	}

	// Memory for device
	checkCUDA (hipMalloc ((void**)&devPtrX, N * sizeof (float)));
	checkCUDA (hipMalloc ((void**)&devPtrY, N * sizeof (float)));

	// Init cuBLAS
	checkCUBLAS (hipblasCreate (&handle));

	// Memcpy host to device
	checkCUBLAS (hipblasSetVector (N, sizeof (float), x, 1, devPtrX, 1));
	checkCUBLAS (hipblasSetVector (N, sizeof (float), y, 1, devPtrY, 1));

	// Saxpy with GPU
	gettimeofday(&start, NULL);
	checkCUBLAS (hipblasSaxpy (handle, N, &alpha, devPtrX, incx, devPtrY, incy));
	gettimeofday(&end, NULL);

	// Print duration
	ms = getMillisecond(start, end);
	printf("GPU time: %f (ms)\n", ms);

	// Memcpy device to host
	checkCUBLAS (hipblasGetVector (N, sizeof (float), devPtrY, 1, resultGPU, 1));

	// Saxpy with CPU
	gettimeofday(&start, NULL);
	for (int i = 0; i < N; i++) {
		resultCPU[i] = x[i]*alpha + y[i];
	}
	gettimeofday(&end, NULL);

	// Print duration
	ms = getMillisecond(start, end);
	printf("CPU time: %f (ms)\n", ms);

	// Validate the result
	float error = 0;
	for (int i = 0; i < N; i++) {
		error += abs((resultCPU[i] - resultGPU[i]) / resultCPU[i]);
	}
	error = error / N * 100;
	printf ("Mean Absolute Percentage Error: %f (%%)\n", error);

	// Free
	checkCUDA (hipFree (devPtrX));
	checkCUDA (hipFree (devPtrY));
	checkCUBLAS (hipblasDestroy (handle));
	free(x);
	free(y);
	free(resultCPU);
	free(resultGPU);
	return EXIT_SUCCESS;
}
