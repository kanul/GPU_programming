#include "hip/hip_runtime.h"
#include "rbm.h"
#include "Timer.h"
#include <sys/time.h>

float *d_v1;
float *d_v2;
float *d_h1;
float *d_h2;

float *d_w;
float *d_bias_hidden;
float *d_bias_visible;
float *d_rand;

float *d_weight_g;
float *d_weight_incs_g;
float *d_hidden_bias_g;
float *d_buf_g;
float *d_diff_g;

__device__ inline float _sigmoid(float x) 
{
	return (1.0 / (1.0 + exp(-x))); 
}

// dotproduct between matrix A's column x and matrix B's row y
__device__ inline float dotprod_tr(int x, int y, int len, float *A, float *B, float bias)
{
	float sum = 0;
	for(int i=0; i<len; i++) {
		sum += (A[x*len + i] * B[y*len + i]); 
	}

	return sum + bias;
}

__global__ void matmulGPU_global(int nc, int nv, int nh, float *v, float *w, float *h, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nc && y < nh ) {
		float sum = dotprod_tr(x, y, nv, v, w, bias[y]);
		h[x*nh + y] = sum;
	}
}

__global__ void matmulGPU_sigmoid_global(int nc, int nv, int nh, float *v,float *w, float *h, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nc && y < nh ) {
		float sum = dotprod_tr(x, y, nv, v, w, bias[y]);
		h[x*nh + y] = _sigmoid(sum);
	}
}

__global__ void matmulGPU_exp_global(int nc, int nv, int nh, float *v,float *w, float *h, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nc && y < nh ) {
		float sum = dotprod_tr(x, y, nv, v, w, bias[y]);
		h[x*nh + y] = exp(sum);
	}
}

__global__ void matmulGPU_addnoise_global(int nc, int nv, int nh, float *h,float *w, float *v, float* noise, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nc && y < nh ) {
		float sum = 0;
		for(int i=0; i<nh; i++)
			sum += (h[x * nh + i]+noise[x*nh +i]) * w[i*nv + y];
		sum += bias[y];
		v[x*nv + y] = _sigmoid(sum);
	}
}

__global__ void matmulGPU_bernoulli_global(int nc, int nv, int nh, float *h,float *w, float *v, float* hs, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( x >= nc || y >= nv )
		return;

	float sum = 0;
	for(int i=0; i<nh; i++)
		sum += (hs[x*nh + i] < h[x*nh + i]? 1.0:0.0) * w[i*nv + y];
	sum += bias[y];
	v[x*nv + y] = _sigmoid(sum);
}

// GEMM_NT
__global__ void forward_gpu_sigmoid(int ns, int nh, int nv, float *v, float *w, float *h, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nh && y < ns ) {
		float sum = 0;
		for(int i=0; i<nv; i++) {
			sum += (v[y*nv + i] * w[x*nv + i]);
		}
		sum += bias[x];
		h[y*nh + x] = _sigmoid(sum);
	}
}

// GEMM_NT
__global__ void forward_gpu_exp(int ns, int nh, int nv, float *v, float *w, float *h, float* bias)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nh && y < ns ) {
		float sum = 0;
		for(int i=0; i<nv; i++) {
			sum += (v[y*nv + i] * w[x*nv + i]);
		}
		sum += bias[x];
		h[y*nh + x] = exp(sum);
	}
}

// GEMM_NN
__global__ void gradient_non_linear_gpu(int ns, int nv, int nh, float *hd, float *w, float *vd, float *v)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nv && y < ns ) {
		float sum = 0;
		for(int i=0; i<nh; i++) {
			sum += (hd[y*nh + i] * w[i*nv + x]);
		}
		sum *= v[y*nv + x] * (1.0 - v[y*nv + x]);
		vd[y*nv + x] = sum;
	}
}

// GEMM_NN
__global__ void gradient_linear_gpu(int ns, int nv, int nh, float *hd, float *w, float *vd)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nv && y < ns ) {
		float sum = 0;
		for(int i=0; i<nh; i++) {
			sum += (hd[y*nh + i] * w[i*nv + x]);
		}
		vd[y*nv + x] = sum;
	}
}

// GEMM_TN
__global__ void calc_weight_incs_gpu(int nh, int nv, int ns, float *diff, float *buf, float *weight_incs)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nv && y < nh ) {
		float sum = 0;
		for(int i=0; i<ns; i++) {
			sum += buf[i*nv + x] * diff[i*nh + y];
		}
		weight_incs[y*nv + x] = sum;
	}
}

// sum reduction
__global__ void weight_incs_add_diff_gpu(int ns, int nh, float *diff, float *weight_incs)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < nh && y < ns ) {
		float val = diff[y*nh + x];
		atomicAdd(weight_incs + y, val);
	}
}

__global__ void dot_gpu(int M, int N, float *a, float *b, float *r, float *bias, bool type)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < M)
	{
		float sum = 0;
		for (int i = 0; i < N; i++)
			sum += a[x*N + i] * b[i];
		sum += bias[x];
		if (type == 0)
			sum = exp(sum);
		else
			sum = _sigmoid(sum);
		atomicAdd(r + x, sum);
	}
}

const Vector& bernoulli(const Vector& input, Vector& output)
{ 
	startTimer("##### bernoulli()");
	static std::default_random_engine eng(::time(NULL));
	static std::uniform_real_distribution<float> rng(0.0, 1.0);

	for (size_t i=0; i<input.size(); ++i) { output[i] = (rng(eng) < input[i])? 1.0 : 0.0; } 
	endTimer("##### bernoulli()");
	return output;
}

float sigmoid(float x) 
{ 
	return (1.0 / (1.0 + exp(-x))); 
}

/* 
 * RBM
 */
int RBM::mirror(const RBM& rbm)
{
	size_t n_visible = bias_visible_.size(), n_hidden = bias_hidden_.size();
	if (n_hidden != rbm.num_visible() || n_visible != rbm.num_hidden()) { 
		std::cout << "not mirrorable" << std::endl;
		return -1;
	}

	bias_visible_ = rbm.bias_hidden_;
	bias_hidden_ = rbm.bias_visible_;
	for (size_t i = 0; i < n_visible; ++i) {
		for (size_t j = 0; j < n_hidden; ++j) {
			weight_[j * n_visible + i] = rbm.weight_[i * n_hidden + j];
		}
	}
	return 0;  
}

const Vector& RBM::activate_visible(const Vector& hidden, Vector& visible) const
{
	startTimer("##### activate_visible()");
	size_t n_visible = bias_visible_.size(), n_hidden = bias_hidden_.size();

	std::fill(visible.begin(), visible.end(), 0);
	for (size_t i = 0; i < n_visible; ++i) {
		float s = 0;
		for (size_t j = 0; j < n_hidden; ++j) 
			s += hidden[j] * weight_[j * n_visible+ i];
		s += bias_visible_[i];

		s = sigmoid(s);
		visible[i] = s;
	}
	endTimer("##### activate_visible()");

	return visible;
}

#define CPU_GEMM
float RBM::train(Batch inputs, const Conf& conf)
{
	struct timeval t_start, t_end, t_timer;
	gettimeofday(&t_start, NULL);
	size_t n_samples = inputs.size();
	size_t n_visible = bias_visible_.size(), n_hidden = bias_hidden_.size();
	float momentum = conf.momentum_, learning_rate = conf.learning_rate_, weight_cost = conf.weight_cost_;

	startTimer("### RBM train()");
	startTimer("#### RBM train()-delta");

	// temporary results
	Vector v1(n_visible), h1(n_hidden), v2(n_visible), h2(n_hidden), hs(n_hidden);

	//delta
	Vector gw(n_visible * n_hidden), gv(n_visible), gh(n_hidden);
#if defined CPU_GEMM
	static std::default_random_engine eng(::time(NULL));
	static std::normal_distribution<float> rng(0.0, 1.0);
	static std::uniform_real_distribution<float> uni(0.0, 1.0);
	float *h_v1 = new float[n_visible * n_samples];
	float *h_v2 = new float[n_visible * n_samples];
	float *h_h1 = new float[n_hidden * n_samples];
	float *h_h2 = new float[n_hidden * n_samples];

	int ofs = 0;
	for (auto const& input: inputs) {
		std::copy (input.begin(), input.end(), h_v1 + ofs*n_visible);
		ofs++;
	}

	// activate_hidden(v1, h1)
	// (nc x n_vis) * (n_vis x n_hid) = (nc * n_hid)
	for (int c = 0 ; c < n_samples ; ++c)
	{
		for (int i =0 ; i < n_hidden ; ++i)
		{
			float sum = 0;
			for (int j = 0 ; j < n_visible ; ++j)
				sum += h_v1[c*n_visible +j] * weight_[i*n_visible + j];
			sum += bias_hidden_[i];
			if (type_ == Type::SIGMOID) sum = sigmoid(sum);
			else if (type_ == Type::EXP) sum = exp(sum);
			h_h1[c*n_hidden +i] = sum;
		}
	}

	// activate_visible(h1, v2)
	// (nc x n_hid) * (n_hid x n_vis) = (nc * n_vis)
	if (type_ == Type::LINEAR)
		for (int i = 0 ; i < n_samples*n_hidden ; ++i)
			h_h1[i] += rng(eng);
	else
		for (int i = 0 ; i < n_samples*n_hidden ; ++i)
			h_h1[i] = uni(eng) < h_h1[i]? 1.0 : 0.0;

	for (int c = 0 ; c < n_samples ; ++c)
	{
		for (int i = 0 ; i < n_visible ; ++i)
		{
			float sum = 0;
			for (int j = 0 ; j < n_hidden ; ++j)
				sum += h_h1[c*n_hidden +j] * weight_[j*n_visible + i];
			sum += bias_visible_[i];
			sum = sigmoid(sum);
			h_v2[c*n_visible + i] = sum;
		}
	}

	// activate_hidden(v2, h2)
	// (nc x n_vis) * (n_vis x n_hid) = (nc * n_hid)
	for (int c = 0 ; c < n_samples ; ++c)
	{
		for (int i = 0 ; i < n_hidden ; ++i)
		{
			float sum = 0;
			for (int j = 0 ; j < n_visible ; ++j)
				sum += h_v2[c*n_visible + j] * weight_[i*n_visible + j];
			sum += bias_hidden_[i];
			if (type_ == Type::SIGMOID) sum = sigmoid(sum);
			else if (type_ == Type::EXP) sum = exp(sum);
			h_h2[c*n_hidden +i] = sum;
		}
	}

	for (int c = 0 ; c < n_samples ; ++c)
		for (size_t i = 0; i < n_visible; ++i)
			for (size_t j = 0; j < n_hidden; ++j) 
				gw[j*n_visible + i] += h_h1[c*n_hidden + j] * h_v1[c*n_visible + i] - h_h2[c*n_hidden + j] * h_v2[c*n_visible + i];

	for (int c = 0 ; c < n_samples ; ++c)
		for (int i = 0 ; i < n_hidden ; ++i)
			gh[i] += h_h1[c*n_hidden + i] - h_h2[c*n_hidden + i];

	for (int c = 0 ; c < n_samples ; ++c)
		for (int i = 0 ; i < n_visible ; ++i)
			gv[i] += h_v1[c*n_visible + i] - h_v2[c*n_visible + i];

	delete[] h_v1;
	delete[] h_v2;
	delete[] h_h1;
	delete[] h_h2;
#else
	for (auto const& input: inputs) {
		v1 = input;
		this->activate_hidden(v1, h1);
		this->activate_visible((type_ == Type::LINEAR? add_noise(h1, hs): bernoulli(h1, hs)), v2);
		this->activate_hidden(v2, h2);

		for (size_t i = 0; i < n_visible; ++i) {
			for (size_t j = 0; j < n_hidden; ++j) 
				gw[j * n_visible + i] += h1[j] * v1[i] - h2[j] * v2[i];
		}

		//      gh += (h1 - h2);
		//      gv += (v1 - v2);
		v::saxpy2(gh, 1.0, h1, -1.0, h2);
		v::saxpy2(gv, 1.0, v1, -1.0, v2);
	}
#endif // CPU_GEMM

	endTimer("#### RBM train()-delta");
	startTimer("#### RBM train()-update");

	//update
	//    gw /= float(n_samples);
	//    gw -= weight_ * weight_cost;
	v::saxpy(1.0/n_samples, gw, -weight_cost, weight_);
	//    weight_inc_ = weight_inc_ * momentum + gw * learning_rate;
	v::saxpy(momentum, weight_inc_, learning_rate, gw);

	//    weight_ += weight_inc_;
	v::saxpy(weight_, 1.0, weight_inc_);

	//    gh /= float(n_samples); 
	//    bias_hidden_inc_ = bias_hidden_inc_ * momentum + gh * learning_rate;
	v::saxpy(momentum, bias_hidden_inc_, learning_rate / n_samples, gh);
	//    bias_hidden_ += bias_hidden_inc_;
	v::saxpy(bias_hidden_, 1.0, bias_hidden_inc_);

	//    gv /= float(n_samples); 
	//    bias_visible_inc_ = bias_visible_inc_ * momentum + gv * learning_rate;
	v::saxpy(momentum, bias_visible_inc_, learning_rate / n_samples, gv);
	//    bias_visible_ += bias_visible_inc_;
	v::saxpy(bias_visible_, 1.0, bias_visible_inc_);

	//    float error = sqrt(gv.dot(gv) / n_visible);
	v::scale(gv, 1.0/n_samples);
	float error = sqrt(v::dot(gv, gv) / n_visible);
	//    std::cout << "error: " << error << ", energy: " << free_energy() << std::endl;
	gettimeofday(&t_end, NULL);
	timersub(&t_end, &t_start, &t_timer);
	printf("time = %lf\n", t_timer.tv_sec * 1000.0 +  t_timer.tv_usec / 1000.0);

	endTimer("#### RBM train()-update");
	endTimer("### RBM train()");
	return error;
}

float RBM::train_gpu(Batch inputs, const Conf& conf)
{
	size_t n_samples = inputs.size();
	size_t n_visible = bias_visible_.size(), n_hidden = bias_hidden_.size();
	float momentum = conf.momentum_, learning_rate = conf.learning_rate_, weight_cost = conf.weight_cost_;

	startTimer("### RBM train()");
	startTimer("#### RBM train()-delta");

	// temporary results
	Vector v1(n_visible), h1(n_hidden), v2(n_visible), h2(n_hidden), hs(n_hidden);

	//delta
	Vector gw(n_visible * n_hidden), gv(n_visible), gh(n_hidden);
	static std::default_random_engine eng(::time(NULL));
	static std::normal_distribution<float> rng(0.0, 1.0);
	static std::uniform_real_distribution<float> uni(0.0, 1.0);
	float *h_v1 = new float[n_visible * n_samples];
	float *h_v2 = new float[n_visible * n_samples];
	float *h_h1 = new float[n_hidden * n_samples];
	float *h_h2 = new float[n_hidden * n_samples];
	float *h_rand = new float[n_hidden * n_samples];

	int ofs = 0;
	for (auto const& input: inputs) {
		std::copy (input.begin(), input.end(), h_v1 + ofs*n_visible);
		ofs++;
	}

	hipMemcpy(d_v1, h_v1, n_samples * n_visible * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_w, weight_.data(), n_visible * n_hidden * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy(d_bias_hidden, bias_hidden_.data(), n_hidden * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy(d_bias_visible, bias_visible_.data(), n_visible * sizeof (float), hipMemcpyHostToDevice);

	int threads_width=16;
	int d_x = (n_samples % threads_width) ? (n_samples/threads_width+1) : (n_samples/threads_width);
	int d_y = (n_hidden % threads_width) ? (n_hidden/threads_width+1) : (n_hidden/threads_width);
	dim3 dim_threads(threads_width, threads_width);
	dim3 dim_grid(d_x, d_y);

	if (type_ == Type::SIGMOID)
		matmulGPU_sigmoid_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v1, d_w, d_h1, d_bias_hidden);
	else if (type_ == Type::EXP)
		matmulGPU_exp_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v1, d_w, d_h1, d_bias_hidden);
	else
		matmulGPU_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v1, d_w, d_h1, d_bias_hidden);
	//    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

	// cuRAND?
	if (type_ == Type::LINEAR)
		for (int i = 0 ; i < n_samples*n_hidden ; ++i)
			h_rand[i] = rng(eng);
	else
		for (int i = 0 ; i < n_samples*n_hidden ; ++i)
			h_rand[i] = uni(eng);

	hipMemcpy(d_rand, h_rand, n_samples*n_hidden*sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	d_y = n_visible % threads_width ? n_visible / threads_width + 1 : n_visible / threads_width;
	dim_threads = dim3(threads_width, threads_width);
	dim_grid = dim3(d_x, d_y);

	if (type_ == Type::LINEAR)
		matmulGPU_addnoise_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_h1, d_w, d_v2, d_rand, d_bias_visible);
	else
		matmulGPU_bernoulli_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_h1, d_w, d_v2, d_rand, d_bias_visible);

	hipDeviceSynchronize();

	d_y = n_hidden % threads_width ? n_hidden / threads_width + 1 : n_hidden / threads_width;
	dim_threads = dim3(threads_width, threads_width);
	dim_grid = dim3(d_x, d_y);

	if (type_ == Type::SIGMOID)
		matmulGPU_sigmoid_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v2, d_w, d_h2, d_bias_hidden);
	else if (type_ == Type::EXP)
		matmulGPU_exp_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v2, d_w, d_h2, d_bias_hidden);
	else 
		matmulGPU_global<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
				d_v2, d_w, d_h2, d_bias_hidden);

	hipDeviceSynchronize();

	hipMemcpy(h_h1, d_h1, n_hidden * n_samples * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_h2, d_h2, n_hidden * n_samples * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_v2, d_v2, n_visible * n_samples * sizeof(float), hipMemcpyDeviceToHost);

	for (int c = 0 ; c < n_samples ; ++c)
		for (size_t i = 0; i < n_visible; ++i)
			for (size_t j = 0; j < n_hidden; ++j) 
				gw[j*n_visible + i] += h_h1[c*n_hidden + j] * h_v1[c*n_visible + i] - h_h2[c*n_hidden + j] * h_v2[c*n_visible + i];

	for (int c = 0 ; c < n_samples ; ++c)
		for (int i = 0 ; i < n_hidden ; ++i)
			gh[i] += h_h1[c*n_hidden + i] - h_h2[c*n_hidden + i];

	for (int c = 0 ; c < n_samples ; ++c)
		for (int i = 0 ; i < n_visible ; ++i)
			gv[i] += h_v1[c*n_visible + i] - h_v2[c*n_visible + i];

	delete[] h_v1;
	delete[] h_v2;
	delete[] h_h1;
	delete[] h_h2;
	delete[] h_rand;


	endTimer("#### RBM train()-delta");
	startTimer("#### RBM train()-update");

	//update
	//    gw /= float(n_samples);
	//    gw -= weight_ * weight_cost;
	v::saxpy(1.0/n_samples, gw, -weight_cost, weight_);
	//    weight_inc_ = weight_inc_ * momentum + gw * learning_rate;
	v::saxpy(momentum, weight_inc_, learning_rate, gw);

	//    weight_ += weight_inc_;
	v::saxpy(weight_, 1.0, weight_inc_);

	//    gh /= float(n_samples); 
	//    bias_hidden_inc_ = bias_hidden_inc_ * momentum + gh * learning_rate;
	v::saxpy(momentum, bias_hidden_inc_, learning_rate / n_samples, gh);
	//    bias_hidden_ += bias_hidden_inc_;
	v::saxpy(bias_hidden_, 1.0, bias_hidden_inc_);

	//    gv /= float(n_samples); 
	//    bias_visible_inc_ = bias_visible_inc_ * momentum + gv * learning_rate;
	v::saxpy(momentum, bias_visible_inc_, learning_rate / n_samples, gv);
	//    bias_visible_ += bias_visible_inc_;
	v::saxpy(bias_visible_, 1.0, bias_visible_inc_);

	//    float error = sqrt(gv.dot(gv) / n_visible);
	v::scale(gv, 1.0/n_samples);
	float error = sqrt(v::dot(gv, gv) / n_visible);
	//    std::cout << "error: " << error << ", energy: " << free_energy() << std::endl;

	endTimer("#### RBM train()-update");
	endTimer("### RBM train()");
	return error;
}


/* 
 * Layered RBM
 */

int LRBM::build(const std::vector<int>& layers, const std::vector<int>& adjust /* = std::vector<int>() */)
{
	startTimer("## build");
	if (layers.size() <= 1) return -1;

	for (size_t i=0; i<layers.size() - 1; ++i) {
		int n_visible= layers[i] + (adjust.empty()? 0: adjust[i]);
		int n_hidden = layers[i+1];

		max_neurons = std::max(max_neurons, std::max(n_visible, n_hidden));
		max_n_visible = std::max(max_n_visible, n_visible);
		max_n_hidden = std::max(max_n_hidden, n_hidden);
		std::cout << "New RBM " << n_visible << " -> " << n_hidden << std::endl;
		rbms_.push_back(std::unique_ptr<RBM>(new RBM(n_visible, n_hidden)));
	}
	std::cout << "max_neurons: " << max_neurons << std::endl;
	endTimer("## build");

	return 0;
}

std::vector<int> LRBM::offsets(int start) const
{
	int n_layers = rbms_.size() - start;
	std::vector<int> dims(n_layers + 1);
	dims[0] = 0;
	int total = 0;
	for(size_t i=0; i<n_layers; ++i) {
		total += (rbms_[i + start]->num_visible() + 1) * rbms_[i + start]->num_hidden();
		dims[i+1] = total;
	}
	return dims;
}

void LRBM::to_image(Vector& image, int& width, int& height)
{
	width = 0; height = 0;
	auto& rbms = this->rbms_;
	for (auto& rbm: rbms) {
		if (width < rbm->num_hidden() + 1) width = rbm->num_hidden() + 1;
		height += (rbm->num_visible() + 2);  
	}
	image.resize(width * height);

	size_t y_offset = 0;
	for (auto& rbm: rbms) {
		size_t n_visible = rbm->num_visible();
		size_t n_hidden = rbm->num_hidden();
		size_t x_offset = (width - n_hidden) / 2;

		for (size_t j=0; j<n_hidden; ++j)
			image[y_offset * width + x_offset + j] = rbm->bias_hidden_[j];
		for (size_t i=0; i<n_visible; ++i) {
			for (size_t j=0; j<n_hidden; ++j)
				//          image[(y_offset + i) * width + x_offset + j] = rbm->weight_[i * n_hidden + j];
				image[(y_offset + i) * width + x_offset + j] = rbm->weight_[j * n_visible+ i];
			image[(y_offset + i) * width + x_offset + n_hidden] = rbm->bias_visible_[i];
		}
		y_offset += n_visible + 2;
	}
}

void LRBM::store(std::ostream& os) const
{
	int32_t count = rbms_.size();
	os.write(reinterpret_cast<char *>(&count), sizeof(count));
	for (auto const& rbm: rbms_) rbm->store(os);
}

void LRBM::load(std::istream& is)
{
	int32_t count = 0;
	is.read(reinterpret_cast<char *>(&count), sizeof(count));

	rbms_.clear();
	for (size_t i = 0; i < count; ++i) 
	{
		RBMP rbm(new RBM());
		rbm->load(is);
		rbms_.push_back(std::move(rbm));
	}
}

/* 
 * Deep Belief Nets
 */

int DeepBeliefNet::train(std::vector<Vector>& inputs, std::vector<Vector>& labels, 
		int max_layer, LRBM::Conf& conf, bool is_cuda /*= false*/)
{
	startTimer("## DBN train()");
	int n_samples = inputs.size(), n_labels = labels.size();
	if (n_labels > 0 && n_samples != n_labels) {
		std::cerr << "# inputs does not match # labels" << std::endl;
		return -1;
	}

	int max_epoch = conf.max_epoch_, batch_size = conf.batch_size_; 
	int max_batches = std::min(conf.max_batches_, n_samples / batch_size); 

	hipMalloc((void **) &d_v1, max_n_visible * conf.batch_size_ * sizeof(float) );
	hipMalloc((void **) &d_v2, max_n_visible * conf.batch_size_ * sizeof(float) );
	hipMalloc((void **) &d_h1, max_n_hidden * conf.batch_size_ * sizeof(float) );
	hipMalloc((void **) &d_h2, max_n_hidden * conf.batch_size_ * sizeof(float));

	hipMalloc((void **) &d_w, max_n_hidden * max_n_visible * sizeof(float) );

	hipMalloc((void **) &d_bias_hidden, max_n_hidden * sizeof(float));
	hipMalloc((void **) &d_bias_visible, max_n_visible * sizeof(float));

	hipMalloc((void **) &d_rand, max_n_hidden * conf.batch_size_ * sizeof(float));

	std::vector<Vector> probs(n_samples);

	for(int layer = 0; layer < max_layer; ++layer) {
		auto& rbm = this->rbms_[layer];
		RBM::Conf conf;
		//XXX: more epochs and lower learning rate for linear rbm
		if (rbm->type_ == RBM::Type::LINEAR) { max_epoch = 100; conf.learning_rate_ = 0.001; }

		for (int epoch = 0; epoch < max_epoch; ++epoch) {

			//XXX: update momentum
			if (epoch > 5) conf.momentum_ = .9f;


			for (size_t batch = 0; batch < max_batches; ++batch) {
				int start = batch * batch_size, end = std::min(start + batch_size, n_samples);

				Batch data;
				if (layer == 0) 
					data = Batch{inputs.begin() + start, inputs.begin() + end};
				else 
					data = Batch{probs.begin() + start, probs.begin() + end};

				float error;
				if (is_cuda)
					error = rbm->train_gpu(data, conf);
				else
					error = rbm->train(data, conf);

				if ((batch + 1) % 10 == 0) {
					std::cout << "layer: " << layer << ", epoch: " << epoch << ", batch: " << batch + 1 
						<< ", error: " << error << ", energy: " << this->free_energy() << std::endl;
				}

				//save outputs to probs at last epoch
				if (epoch == max_epoch - 1) {
					auto it = data.begin();
					for(int i = start; i < end; ++i) {
						Vector output(rbm->num_hidden());
						rbm->activate_hidden(*it++, output);
						output.swap(probs[i]);
					}

					//attach labels for last layer
					if (layer > 0 && layer + 1 == max_layer - 1 && !labels.empty()) {
						size_t input_size = probs[start].size(), label_size = labels.front().size();
						for (size_t i = start; i < end; ++i) {
							const Vector& label = labels[i];
							Vector& input = probs[i];
							input.resize(input_size + label_size);
							std::copy(label.begin(), label.end(), input.begin() + input_size);
						}
					}
				} // save output 
			} // batch  
		} // epoch
	} // layer

	endTimer("## DBN train()");
	return 0;
}

int DeepBeliefNet::predict(const Vector& sample, Vector& output, Vector& probs)
{
	static std::default_random_engine eng(::time(NULL));
	std::uniform_real_distribution<float> rng(0.0, 1.0);

	Vector input(sample);
	int n_layers = rbms_.size();
	for (int i =0; i<n_layers - 1; ++i) {
		const RBMP& rbm = rbms_[i];
		size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();

		Vector next(n_hidden);
		rbm->activate_hidden(input, next); 
		input.swap(next);  
	}

	RBMP& rbm = rbms_[n_layers - 1];
	size_t n_visible = rbm->num_visible();
	size_t n_hidden = rbm->num_hidden();
	size_t n_input = input.size();
	if (n_input  + output.size() != n_visible) {
		return -1;
	}

	// attach zero-ed labels
	if (n_visible > n_input) input.resize(n_visible);

	Vector h1(n_hidden);
	rbm->activate_hidden(input, h1);

	if (! probs.empty()) 
		probs = h1;

	if (! output.empty()) {
		Vector hs(n_hidden), v2(n_visible);
		rbm->activate_visible(bernoulli(h1, hs), v2);
		std::copy(v2.begin() + n_input, v2.end(), output.begin());
	}

	return 0;    
}

int DeepBeliefNet::gradient(GradientContext& ctx, const Vector& weights, Vector& weight_incs, float& cost)
{
	// GradientConext has input, output data of each layer and information about max_iteration, epoch, and start_layer
	Batch& inputs = ctx.inputs_;
	std::vector<std::vector<Vector>>& probs = ctx.probs_; 
	bool has_targets = !ctx.targets_.empty();

	int max_layer = this->rbms_.size();

	size_t n_hidden = rbms_.back()->num_hidden(); // last layer's number of hidden node
	size_t n_samples = inputs.size();
	std::vector<Vector> diffs(n_samples);

	auto cstart = std::chrono::high_resolution_clock::now();
	auto dims = this->offsets(ctx.start_layer_); // Weight offset of each layer

	startTimer("#### gradient()");
	startTimer("##### gradient()-input forwarding");
	// Initialize weight incs to zero
	std::fill(weight_incs.begin(), weight_incs.end(), 0);
	cost = 0;
	float error = 0;

	// *******************
	// We should calculate ∂E_l/∂w_l and activation function y_l+1 = sigmoid(out_l)
	// ∂E/∂w_l (weight_incs[l]) = ∂E/∂y_l (h_diff[l]) * ∂y_l/∂w_l (h_buf[l])
	// ∂E/∂y_l (h_diff[l]) = ∂E/∂y_l+1 (h_diff[l+1]) * ∂y_l+1/∂y_l
	// ∂y_l+1/∂y_l = ∂y_l+1/∂out_l * ∂out_l/∂y_l = sigmoid(out_l) * (1 - sigmoid(out_l)) * w_l
	// *******************

#if defined CPU_GEMM
	// In this program, value of visible node, hidden node, and etc. is managed by vector type.
	// So first, we calculate the largest size of all layers, then create a buffer based on the maximum size
	for (int layer = 0 ; layer < max_layer ; ++layer)
	{
		int nvis = this->rbms_[layer]->num_visible();
		int nhid = this->rbms_[layer]->num_hidden();

		max_neurons = std::max(max_neurons, std::max(nvis, nhid)); // maximum node size include visible and hidden node
		max_n_visible = std::max(max_n_visible, nvis); // maximum size of visible node
		max_n_hidden = std::max(max_n_hidden, nhid); // maximum size of hidden node
	}

	// TODO: use static qualifier to avoid allocation repeatedly
	float *h_buf = new float[max_neurons * n_samples * (max_layer+1)]; // input data + forward resulti (probs) of each layer
	float *h_weight = new float[max_n_visible * max_n_hidden * max_layer];
	float *h_hidden_bias = new float[max_n_hidden * max_layer];
	float *h_diff = new float[max_neurons * n_samples * max_layer];

	// we mamage data of all layers by 1-d array, so we need to know of offset of each layer
	int ofs_w[max_layer+1]; // offset of each layer for h_weight ex. h_weight[ofs_w[layer] + i]
	int ofs_h[max_layer+1]; // offset of each layer for h_hidden_bias
	int ofs_i[max_layer+1]; // offset of each layer for h_buf when used for input
	int ofs_o[max_layer+1]; // offset of each layer for h_buf when used for output
	int ofs_di[max_layer+1]; // offset of each layer for h_diff when used for input
	int ofs_do[max_layer+1]; // offset of each layer for h_diff when used for output

	ofs_w[0] = ofs_h[0] = ofs_i[0] = 0;
	ofs_o[0] = n_samples * rbms_[0]->num_visible();

	// Copy data in vector of each layer to array, and calculate offset of each layer for weight, bias, input, and output
	for (int i = 1; i <= max_layer; ++i) {
		const RBMP& rbm = this->rbms_[i-1];
		ofs_w[i] = ofs_w[i-1] + rbm->num_weight();
		ofs_h[i] = ofs_h[i-1] + rbm->num_hidden();

		// TODO: weights update frequently, so copy them to h_weight is necessary
		// Copy data in vector of each layer to array
		if (i == 1 || weights.empty()) {
			std::copy (rbm->weight_.begin(), rbm->weight_.end(), &h_weight[ofs_w[i-1]]);
			std::copy (rbm->bias_hidden_.begin(), rbm->bias_hidden_.end(), &h_hidden_bias[ofs_h[i-1]]);
		} else {
			const float *start = weights.data() + dims[i-1];
			const float *end = start + rbm->num_weight();
			memcpy(&h_weight[ofs_w[i-1]], start, (end-start) * sizeof(float));
			start = end;
			end = start + rbm->num_hidden();
			memcpy(&h_hidden_bias[ofs_h[i-1]], start, (end-start) * sizeof(float));
		}

		ofs_i[i] = ofs_o[i-1];
		ofs_o[i] = ofs_i[i] + n_samples * rbm->num_hidden();
	}

	// Calculate offset of each layer for diff data
	ofs_di[max_layer-1] = 0;
	ofs_do[max_layer-1] = n_samples * rbms_.back()->num_hidden();
	for (int i = max_layer-2 ; i >= 0 ; --i) {
		const RBMP& rbm = this->rbms_[i];
		ofs_di[i] = ofs_do[i+1];
		ofs_do[i] = ofs_di[i] + n_samples * rbm->num_visible();
	}

	// copy input data in vector to array
	int ofs = 0;
	for (auto const& input: inputs) {
		std::copy (input.begin(), input.end(), h_buf + ofs*input.size());
		ofs++;
	}

	// Forward each layer ([hidden node] = [visible node] x [weight] + [hidden])
	// It is same as activate_hidden()
	for (int layer=0; layer < max_layer; ++layer) {
		const RBMP& rbm = this->rbms_[layer];
		size_t n_visible = rbm->num_visible();
		n_hidden = rbm->num_hidden();

		for (size_t c = 0; c < n_samples; ++c) 
		{ 
			for (int i = 0 ; i < n_hidden ; ++i)
			{
				float sum = 0;
				for (int j = 0 ; j < n_visible ; ++j)
					sum += h_buf[ofs_i[layer] + c*n_visible +j] * h_weight[ofs_w[layer] + i*n_visible + j];
				sum += h_hidden_bias[ofs_h[layer] + i];
				if (rbm->type_ == RBM::Type::SIGMOID) sum = sigmoid(sum);
				else if (rbm->type_ == RBM::Type::EXP) sum = exp(sum); // Last layer's activation fuction
				h_buf[ofs_o[layer] + c*n_hidden + i] = sum;
			}
		} // sample loop
	} // layer loop

	// For all samples in batch, accumulate cost and error
	// and calculate diffrence between layer's output (here, last layer) and target
	for (size_t sample = 0; sample < n_samples; ++sample) { 
		float *result = &h_buf[ofs_o[max_layer-1] + sample * n_hidden];
		float *diff = &h_diff[sample * n_hidden];
		float s = 0;

		// sum of all hidden node output
		for (int i = 0 ; i < n_hidden ; ++i)
			s += result[i];
		// and normalize result of each hidden node to 0 ~ 1 value
		for (int i = 0 ; i < n_hidden ; ++i)
			result[i] *= (1.0/s);

		const Vector& target = ctx.targets_[sample];
		for(size_t i=0 ; i < n_hidden ; ++i) {
			diff[i] = (result[i] - target[i]); // 
			cost += target[i] * log(result[i]); // Cost Fuction y*log(E(x))
			error += diff[i] * diff[i]; // Mean Square Error (MSE)
		}
	}

	delete[] h_hidden_bias;
#else
	for (size_t sample = 0; sample < n_samples; ++sample) { 
		const Vector& input = inputs[sample];
		v::LightVector bias_hidden, weight;
		for (int layer=0; layer < max_layer; ++layer) {
			const RBMP& rbm = this->rbms_[layer];
			if (layer < ctx.start_layer_ || weights.empty()) { 
				float *start = const_cast<float *>(rbm->weight_.data()), *end = start + rbm->num_weight();
				weight = v::LightVector(start, end);
				start = const_cast<float *>(rbm->bias_hidden_.data()); end = start + rbm->num_hidden();
				bias_hidden = v::LightVector(start, end);
			} else { 
				float *start = const_cast<float *>(weights.data()) + dims[layer - ctx.start_layer_], *end = start + rbm->num_weight();
				weight = v::LightVector(start, end);
				start = end; end = start + rbm->num_hidden();
				bias_hidden = v::LightVector(start, end);
			}

			Vector& output = probs[layer][sample];
			const Vector& _input = (layer == 0? input: probs[layer - 1][sample]);
			RBM::activate_hidden(_input, output, bias_hidden, weight, rbm->type_);
		}

		//output
		Vector& result = probs[max_layer - 1][sample]; 
		Vector& diff = diffs[sample];
		diff.resize(n_hidden);

		if (has_targets) {
			float s = std::accumulate(result.begin(), result.end(), 0.0);
			v::scale(result, 1.0/s);

			const Vector& target = ctx.targets_[sample];
			for(size_t i=0; i<n_hidden; ++i) {
				diff[i] = (result[i] - target[i]);
				cost += target[i] * log(result[i]); 
				error += diff[i] * diff[i];
			}
		} else {
			for(size_t i=0; i<n_hidden; ++i) {
				diff[i] = (result[i] - input[i]) / n_samples;
				cost += input[i] * log(result[i]) + (1 - input[i]) * log(1 - result[i]);  
				error += (result[i] - input[i]) * (result[i] - input[i]);
			}
		}
	}
#endif

	cost = -cost;
	if (! has_targets) cost *= 1/ n_samples;

	endTimer("##### gradient()-input forwarding");
	startTimer("##### gradient()-calc gradient");

	//calculate gradient
#if defined CPU_GEMM
	// weight_incs.size() comes from dims which is created from LRBM::offsets()
	// (accumulation of (num_visible+1)*num_hidden from each layer)
	// TODO: create h_weight_incs for CUDA kernel
	// We should calculate ∂E_l/∂w_l to update each layer's weight
	// Here, ∂E/∂w_l = ∂E/∂y_l * ∂y_l/∂w_l, and ∂E/∂y_l = ∂E/∂y_l+1 * ∂y_l+1/∂y_l
	// We can get ∂E/∂y_l+1 from previous layer,
	for (int layer=max_layer - 1; layer >= 0; --layer) {
		if (layer < ctx.start_layer_) 
			break;

		// y_l+1 = sigmoid(out_l)
		// First, we should calculate ∂E/∂y_l = ∂E/∂y_l+1 * ∂y_l+1/∂y_l.
		// Here ∂y_l+1/∂y_l = ∂y_l+1/∂out_l * ∂out_l/∂y_l = sigmoid(out_l) * (1 - sigmoid(out_l)) * W_l
		// and ∂E/∂y_l+1 is previous output (h_diff[layer + 1])
		if (layer != max_layer - 1) {
			const RBMP& rbm = this->rbms_[layer + 1];
			size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();

			for (size_t sample = 0; sample < n_samples; ++sample) { 
				for (size_t j=0; j<n_visible; ++j) {
					float s = 0;
					for (size_t k=0; k<n_hidden; ++k) {
						s += h_diff[ofs_di[layer+1] + sample*n_hidden + k] * h_weight[ofs_w[layer+1] + k * n_visible + j];
					}
					if (rbms_[layer]->type_ != RBM::Type::LINEAR) // derivative of sigmoid y' = y * (1 - y)
						s *= h_buf[ofs_o[layer] + sample*n_visible + j] * (1.0 - h_buf[ofs_o[layer] + sample*n_visible + j]);
					h_diff[ofs_do[layer+1] + sample*n_visible + j] = s;
				}
			}
		}

		RBMP& rbm = this->rbms_[layer];
		size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();
		size_t offset = dims[layer - ctx.start_layer_];

		// weight_incs is ∂E/∂w_l, and it is used to update weight by gradient decent
		// ∂E/∂w_l = ∂E/∂y_l * ∂y_l/∂w_l = h_buf * h_diff
		for (size_t sample = 0; sample < n_samples; ++sample) { 
			for (size_t j=0; j<n_visible; ++j) {
				for (size_t k=0; k<n_hidden; ++k) {
					weight_incs[offset + k * n_visible + j] += 
						h_buf[ofs_i[layer]+ sample*n_visible + j] * h_diff[ofs_di[layer] + sample*n_hidden + k];
				}
			}
			// It is used to update bias (maybe?)
			for (size_t k=0; k<n_hidden; ++k) {
				weight_incs[offset + n_visible * n_hidden + k] += h_diff[ofs_di[layer] + sample*n_hidden + k];
			}
		}
	}

	delete[] h_buf;
	delete[] h_weight;
	delete[] h_diff;
#else
	for (int layer=max_layer - 1; layer >= 0; --layer) {
		if (layer < ctx.start_layer_) 
			break;

		if (layer != max_layer - 1) {
			const RBMP& rbm = this->rbms_[layer + 1];
			//        const Vector& weight = rbm->weight_; 
			size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();
			size_t offset = dims[layer + 1 - ctx.start_layer_];
			v::LightVector weight(const_cast<float *>(weights.data()) + offset, const_cast<float *>(weights.data()) + offset + rbm->num_weight());
			for (size_t sample = 0; sample < n_samples; ++sample) { 
				Vector diff(n_visible);
				for (size_t j=0; j<n_visible; ++j) {
					float s = 0;
					for (size_t k=0; k<n_hidden; ++k) {
						//              s += diffs[sample][k] * weight[j * n_hidden + k];
						s += diffs[sample][k] * weight[k * n_visible + j];
					}
					if (rbms_[layer]->type_ != RBM::Type::LINEAR)
						s *= probs[layer][sample][j] * (1.0 - probs[layer][sample][j]);
					diff[j] = s;
				}
				diffs[sample].swap(diff);
			} 
		}

		RBMP& rbm = this->rbms_[layer];
		size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();
		size_t offset = dims[layer - ctx.start_layer_];

		for (size_t sample = 0; sample < n_samples; ++sample) { 
			const auto& v = (layer > 0? probs[layer-1][sample] : inputs[sample]);
			const auto& d = diffs[sample];
			for (size_t j=0; j<n_visible; ++j) {
				for (size_t k=0; k<n_hidden; ++k) {
					weight_incs[offset + k * n_visible + j] += v[j] * d[k];
				}
			}
			for (size_t k=0; k<n_hidden; ++k) {
				weight_incs[offset + n_visible * n_hidden + k] += d[k];
			}
		}
	}
#endif

	auto cend = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(cend - cstart).count() / 1000.0;
	std::cout << "evaluating: cost=" << cost << ", error=" << error / n_samples << " in " << duration << "ms" << std::endl;

	endTimer("##### gradient()-calc gradient");
	endTimer("#### gradient()");
	return 0;
}

// translate into C++ from matlab code
//    http://learning.eng.cam.ac.uk/carl/code/minimize/minimize.m
int DeepBeliefNet::minimize(GradientContext& ctx, bool is_cuda)
{
	const float INT = 0.1, EXT = 3.0;
	const float SIG = 0.1, RHO = SIG / 2.0, RATIO = 10;
	const int max_iteration = ctx.max_iteration_;

	startTimer("### minimize()");
	// initialize
	float cost = 0;
	auto dims = this->offsets(ctx.start_layer_); 
	Vector weights(dims.back()), weight_incs(dims.back()); 

	{
		auto offset = weights.begin();
		for (size_t i=ctx.start_layer_; i<this->rbms_.size(); ++i) {
			const RBMP& rbm = this->rbms_[i];
			std::copy(rbm->weight_.begin(), rbm->weight_.end(), offset);
			offset += rbm->num_weight();
			std::copy(rbm->bias_hidden_.begin(), rbm->bias_hidden_.end(), offset);
			offset += rbm->num_hidden();
		}
	}

	if (is_cuda)
		this->gradient_gpu(ctx, weights, weight_incs, cost);
	else
		this->gradient(ctx, weights, weight_incs, cost);

	Vector df0(weight_incs);
	Vector s(df0); v::scale(s, -1.0);
	float d0 = -v::dot(s, s), f0 = cost;
	float d3 = 0, x3 = 1.0 / (1 - d0);

	//    std::cout << "d3=" << d3 << ", d0=" << d0 << ",f0=" << f0 << std::endl;

	bool failed = false;
	// line search
	for (int i=0; i<max_iteration; ++i) {
		// extrapolation
		float best_cost = f0;
		Vector best_weights(weights), best_weight_incs(weight_incs);

		float f3 = 0;
		Vector df3(weights.size());

		int M = 20;
		float f1 = 0, x1 = 0, d1 = 0;
		float f2 = 0, x2 = 0, d2 = 0;
		while (true) {
			x2 = 0; f2 = f0; d2 = d0; 
			f3 = f0; df3 = df0;

			while (true) {
				if (M -- < 0) break;

				Vector tmp_weights(weights);
				//          tmp_weights += s * x3;
				v::saxpy(tmp_weights, x3, s);
				if (is_cuda)
					this->gradient_gpu(ctx, tmp_weights, weight_incs, cost);
				else
					this->gradient(ctx, tmp_weights, weight_incs, cost);
				f3 = cost; df3 = weight_incs;
				if (std::isfinite(cost) && v::isfinite(weight_incs)) {
					//found one and save best result if available
					if (f3 < best_cost) {
						best_cost = f3;
						best_weights = tmp_weights;
						best_weight_incs = weight_incs;
					}
					break;
				}

				//back off and retry
				x3 = (x2 + x3) / 2.0;
			}

			// check slope and done extrapolation?
			d3 = v::dot(df3,s);
			if (d3 > SIG*d0 || f3 > f0 + x3*RHO*d0 || M <= 0) break;

			x1 = x2; f1 = f2; d1 = d2;
			x2 = x3; f2 = f3; d2 = d3;  

			// cubic extrapolation
			float dx = x2-x1;
			float A = 6.0*(f1-f2) + 3.0*(d2+d1)*dx;
			float B = 3.0*(f2-f1) - (2.0*d1+d2)*dx;
			x3 = x1-d1*dx*dx/(B+sqrt(B*B-A*d1*dx));

			// keep it in range
			float upper = x2 * EXT, lower = x2 + INT * dx;
			if (!std::isfinite(x3) || x3 < 0 || x3 > upper) x3 = upper;
			else if (x3 < lower) x3 = lower;
		}

		// interpolation
		float f4 = 0, x4 = 0, d4 = 0;
		while ((std::abs(d3) > -SIG*d0 || f3 > f0 + x3*RHO*d0) && M > 0) {
			if (d3 > 0 || f3 > f0+x3*RHO*d0) {
				x4 = x3; f4 = f3; d4 = d3;        
			} else {
				x2 = x3; f2 = f3; d2 = d3;
			}

			float dx = x4 - x2;
			if (f4 > f0) {
				x3 = x2-(0.5*d2*dx*dx)/(f4-f2-d2*dx);  // quadratic interpolation
			} else {
				float A = 6*(f2-f4)/dx+3*(d4+d2);     // cubic interpolation
				float B = 3*(f4-f2)-(2*d2+d4)*dx;
				x3 = x2+(sqrt(B*B-A*d2*dx*dx)-B)/A; 
			}

			if (! std::isfinite(x3)) {
				//          std::cout << "x3 = " << x3 << " not usable" << std::endl;
				x3 = (x2 + x4) / 2;
			}

			// keep it in range
			x3 = std::max(std::min(x3, x4-INT*(x4-x2)),x2+INT*(x4-x2));

			Vector tmp_weights(weights);
			//        tmp_weights += s * x3;
			v::saxpy(tmp_weights, x3, s);
			if (is_cuda)
				this->gradient_gpu(ctx, tmp_weights, weight_incs, cost);
			else
				this->gradient(ctx, tmp_weights, weight_incs, cost);
			f3 = cost; df3 = weight_incs;
			if (f3 < best_cost) {
				best_cost = f3;
				best_weights = tmp_weights;
				best_weight_incs = weight_incs;
			}

			--M;
			//        d3 = df3.dot(s);
			d3 = v::dot(df3,s);
		}

		if (std::abs(d3) < -SIG*d0 && f3 < f0 + x3*RHO*d0) { // succeeded
			//        weights += s * x3; 
			v::saxpy(weights, x3, s);
			f0 = f3; 
			//        s *= (df3.dot(df3) - df3.dot(df0)) / df0.dot(df0); s -= df3; // Polack-Ribiere CG direction
			float g = (v::dot(df3,df3) - v::dot(df3, df0)) / v::dot(df0, df0);
			v::saxpy(g, s, -1.0, df3); // Polack-Ribiere CG direction
			//        d3 = d0; d0  = df3.dot(s); df0 = df3; 
			d3 = d0; d0  = v::dot(df3, s); df0 = df3; 
			if (d0 > 0) {
				//          s = -df0; d0 = -df0.dot(df0);
				s = df0; v::scale(s, -1.0); d0 = -v::dot(df0, df0);
			}

			x3 = x3 * std::min(RATIO, float(d3 / (d0 - 1e-37)));
			failed = false;
			std::cout << "found: iteration i=" << i << ", cost=" << f3 << std::endl;
		} else { // failed
			std::cout << "x3 = " << x3 << " failed" << std::endl;
			weights = best_weights; f0 = best_cost; df0 = best_weight_incs; 
			if (failed) break;  

			//        s = -df0; d0 = - s.dot(s); x3 = 1.0/(1.0-d0);
			s = df0; v::scale(s, -1.0); d0 = -v::dot(s, s); x3 = 1.0/(1.0-d0);
			failed = true;
		}
	}

	//apply the new weights
	{
		auto offset = weights.begin();
		for (size_t i=ctx.start_layer_; i<this->rbms_.size(); ++i) {
			const RBMP& rbm = this->rbms_[i];
			std::copy(offset, offset + rbm->num_weight(), rbm->weight_.begin());
			offset += rbm->num_weight();
			std::copy(offset, offset + rbm->num_hidden(), rbm->bias_hidden_.begin());
			offset += rbm->num_hidden();
		}
	}

	std::cout << "applying new weights to " << ctx.start_layer_ << "+" << std::endl;
	endTimer("### minimize()");
	return 0;
}

int DeepBeliefNet::gradient_gpu(GradientContext& ctx, const Vector& weights, Vector& weight_incs, float& cost)
{
	Batch& inputs = ctx.inputs_;
	std::vector<std::vector<Vector>>& probs = ctx.probs_; 
	bool has_targets = !ctx.targets_.empty();

	int max_layer = this->rbms_.size();

	size_t n_hidden = rbms_.back()->num_hidden();
	size_t n_samples = inputs.size();
	std::vector<Vector> diffs(n_samples);

	auto cstart = std::chrono::high_resolution_clock::now();
	auto dims = this->offsets(ctx.start_layer_);

	startTimer("#### gradient()");
	startTimer("##### gradient()-input forwarding");
	// Initialize weight incs to zero
	std::fill(weight_incs.begin(), weight_incs.end(), 0);
	cost = 0;
	float error = 0;
	// In this program, value of visible node, hidden node, and etc. is managed by vector type.
	// So first, we calculate the largest size of all layers, then create a buffer based on the maximum size
	for (int layer = 0 ; layer < max_layer ; ++layer)
	{
		int nvis = this->rbms_[layer]->num_visible();
		int nhid = this->rbms_[layer]->num_hidden();

		max_neurons = std::max(max_neurons, std::max(nvis, nhid));
		max_n_visible = std::max(max_n_visible, nvis);
		max_n_hidden = std::max(max_n_hidden, nhid);
	}

	// Allocate GPU Memory Buffer
	// Here, we allocate buffer and free that buffer Every time you call gradient_gpu function
	// But, the buffer size is always the same, so it is better to allocate buffer at once before call minimize function
	hipMalloc((void **)&d_buf_g, max_neurons * n_samples * (max_layer + 1) * sizeof(float));
	hipMalloc((void **)&d_weight_g, max_n_visible * max_n_hidden * max_layer * sizeof(float));
	hipMalloc((void **)&d_hidden_bias_g, max_n_hidden * max_layer * sizeof(float));
	hipMalloc((void **)&d_diff_g, max_neurons * n_samples * max_layer * sizeof(float));
	hipMalloc((void **)&d_weight_incs_g, weight_incs.size() * sizeof(float));
	// Initialize weight incs to zero
	hipMemset(d_weight_incs_g, 0, weight_incs.size() * sizeof(float));

	// We need host buffer for each layer's forward output and backward diff data
	// Here, we allocate buffer using the largest size of all layers for convenience
	// h_buf is needed, because this program use vector data type to store data for each layer
	// h_diff is needed, because we will calculate last layer's diff, cost, and error for better performance (data is small)
	float *h_buf = new float[max_neurons * n_samples * (max_layer+1)];
	float *h_diff = new float[max_neurons * n_samples * max_layer];
	int ofs_w[max_layer+1]; // offset of each layer for h_weight ex. h_weight[ofs_w[layer] + i]
	int ofs_h[max_layer+1]; // offset of each layer for h_hidden_bias
	int ofs_i[max_layer+1]; // offset of each layer for h_buf when used for input
	int ofs_o[max_layer+1]; // offset of each layer for h_buf when used for output
	int ofs_di[max_layer+1]; // offset of each layer for h_diff when used for input
	int ofs_do[max_layer+1]; // offset of each layer for h_diff when used for output

	ofs_w[0] = ofs_h[0] = ofs_i[0] = 0;
	ofs_o[0] = n_samples * rbms_[0]->num_visible();

	// Copy weight and bias in vector of each layer to device memory, and calculate offset of each layer for weight, bias, input, and output
	for (int i = 1; i <= max_layer; ++i) {
		const RBMP& rbm = this->rbms_[i-1];
		ofs_w[i] = ofs_w[i-1] + rbm->num_weight();
		ofs_h[i] = ofs_h[i-1] + rbm->num_hidden();

		if (i == 1 || weights.empty()) {
			hipMemcpy(d_weight_g + ofs_w[i-1], &rbm->weight_[0], 
					rbm->weight_.size() * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_hidden_bias_g + ofs_h[i-1], &rbm->bias_hidden_[0], 
					rbm->bias_hidden_.size() * sizeof(float), hipMemcpyHostToDevice);
		} else {
			const float *start = weights.data() + dims[i-1];
			const float *end = start + rbm->num_weight();
			hipMemcpy(d_weight_g + ofs_w[i-1], start, 
					(end-start) * sizeof(float), hipMemcpyHostToDevice);
			start = end;
			end = start + rbm->num_hidden();
			hipMemcpy(d_hidden_bias_g + ofs_h[i-1], start, 
					(end - start) * sizeof(float), hipMemcpyHostToDevice);
		}

		ofs_i[i] = ofs_o[i-1];
		ofs_o[i] = ofs_i[i] + n_samples * rbm->num_hidden();
	}

	ofs_di[max_layer-1] = 0;
	ofs_do[max_layer-1] = n_samples * rbms_.back()->num_hidden();
	for (int i = max_layer-2 ; i >= 0 ; --i) {
		const RBMP& rbm = this->rbms_[i];
		ofs_di[i] = ofs_do[i+1];
		ofs_do[i] = ofs_di[i] + n_samples * rbm->num_visible();
	}

	// Copy input data in vector to array (n_samples * input_size)
	int ofs = 0;
	for (auto const& input: inputs) {
		std::copy (input.begin(), input.end(), h_buf + ofs*input.size());
		ofs++;
	}

	// Copy input data in host memory to device memory
	hipMemcpy(d_buf_g, h_buf, n_samples * inputs.size() * sizeof(float), hipMemcpyHostToDevice);

	// Forward each layer ((output)[n_samples x n_hidden] = (input)[n_samples x n_visible] x (weight)[n_visible x n_hidden] + (bias)[n_hidden])
	// It is same as activate_hidden()
	for (int layer=0; layer < max_layer; ++layer) {
		const RBMP& rbm = this->rbms_[layer];
		size_t n_visible = rbm->num_visible();
		n_hidden = rbm->num_hidden();

		int threads_width=16;
		int d_x = (n_hidden % threads_width) ? (n_hidden/threads_width+1) : (n_hidden/threads_width);
		int d_y = (n_samples % threads_width) ? (n_samples/threads_width+1) : (n_samples/threads_width);
		dim3 dim_threads(threads_width, threads_width);
		dim3 dim_grid(d_x, d_y);

		float *d_input = d_buf_g + ofs_i[layer]; // layer's input
		float *d_output = d_buf_g + ofs_o[layer]; // layer's output
		float *d_bias = d_hidden_bias_g + ofs_h[layer]; // layer's bias (bias of hidden node)
		float *d_weight = d_weight_g + ofs_w[layer]; // layer's weight

		// It is the same as GEMM_NT
		if (rbm->type_ == RBM::Type::SIGMOID)
			forward_gpu_sigmoid<<<dim_grid, dim_threads>>>(n_samples, n_hidden, n_visible, 
					d_input, d_weight, d_output, d_bias);
		else if (rbm->type_ == RBM::Type::EXP) // Expotential function is used in last layer
			forward_gpu_exp<<<dim_grid, dim_threads>>>(n_samples, n_hidden, n_visible, 
					d_input, d_weight, d_output, d_bias);
		hipDeviceSynchronize();

		// Copy output data from device memory to host memory
		// If we caculate last layer's diff, cost, and error in GPU, this step is unnecessary
		hipMemcpy(h_buf + ofs_o[layer], d_output, n_samples * n_hidden * sizeof(float), hipMemcpyDeviceToHost);
	} // layer loop

	// For all samples in batch, accumulate cost and error
	// and calculate diffrence between layer's output (here, last layer) and target in host
	for (size_t sample = 0; sample < n_samples; ++sample) { 
		float *result = &h_buf[ofs_o[max_layer-1] + sample * n_hidden];
		float *diff = &h_diff[sample * n_hidden];
		float s = 0;

		for (int i = 0 ; i < n_hidden ; ++i)
			s += result[i];
		for (int i = 0 ; i < n_hidden ; ++i)
			result[i] *= (1.0/s);

		const Vector& target = ctx.targets_[sample];
		for(size_t i=0 ; i < n_hidden ; ++i) {
			diff[i] = (result[i] - target[i]);
			cost += target[i] * log(result[i]); 
			error += diff[i] * diff[i];
		}
	}

	cost = -cost;
	if (! has_targets) cost *= 1/ n_samples;

	endTimer("##### gradient()-input forwarding");
	startTimer("##### gradient()-calc gradient");

	// Copy last layer's diff from host to device
	hipMemcpy(d_diff_g, h_diff, n_samples * n_hidden * sizeof(float), hipMemcpyHostToDevice);

	//calculate gradient
	// weight_incs.size() comes from dims which is created from LRBM::offsets()
	// (accumulation of (num_visible+1)*num_hidden from each layer)
	// We should calculate ∂E_l/∂w_l to update each layer's weight
	// Here, ∂E/∂w_l = ∂E/∂y_l * ∂y_l/∂w_l, and ∂E/∂y_l = ∂E/∂y_l+1 * ∂y_l+1/∂y_l
	// We can get ∂E/∂_l+1 from previous layer,
	for (int layer=max_layer - 1; layer >= 0; --layer) {
		if (layer < ctx.start_layer_) 
			break;

		if (layer != max_layer - 1) {
			const RBMP& rbm = this->rbms_[layer + 1];
			size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();

			float *d_diff = d_diff_g + ofs_di[layer+1];
			float *d_weight = d_weight_g + ofs_w[layer+1];
			float *d_buf = d_buf_g + ofs_o[layer];
			float *d_diff_out = d_diff_g + ofs_do[layer+1];

			int threads_width=16;
			int d_x = (n_visible % threads_width) ? (n_visible/threads_width+1) : (n_visible/threads_width);
			int d_y = (n_samples % threads_width) ? (n_samples/threads_width+1) : (n_samples/threads_width);
			dim3 dim_threads(threads_width, threads_width);
			dim3 dim_grid(d_x, d_y);

			// Non linear means no activation function
			// Same as GEMM_NN
			if (rbms_[layer]->type_ != RBM::Type::LINEAR)
				gradient_non_linear_gpu<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
						d_diff, d_weight, d_diff_out, d_buf);
			else
				gradient_linear_gpu<<<dim_grid, dim_threads>>>(n_samples, n_visible, n_hidden, 
						d_diff, d_weight, d_diff_out);
			hipDeviceSynchronize();
		}

		// Below code is to calculate weight_incs(∂E/∂w_l) and bias_incs(it is stored weight_incs buffer)
		// weight_incs is ∂E/∂w_l, and it is used to update weight by gradient decent
		RBMP& rbm = this->rbms_[layer];
		size_t n_visible = rbm->num_visible(), n_hidden = rbm->num_hidden();
		size_t offset = dims[layer - ctx.start_layer_];

		float *d_diff = d_diff_g + ofs_di[layer];
		float *d_buf = d_buf_g + ofs_i[layer];
		float *d_weight_incs = d_weight_incs_g + offset;

		int threads_width=16;
		int d_x = (n_visible % threads_width) ? (n_visible/threads_width+1) : (n_visible/threads_width);
		int d_y = (n_hidden % threads_width) ? (n_hidden/threads_width+1) : (n_hidden/threads_width);
		dim3 dim_threads(threads_width, threads_width);
		dim3 dim_grid(d_x, d_y);

		// Same as GEMM_TN
		// ∂E/∂w_l = ∂E/∂y_l * ∂y_l/∂w_l = h_buf * h_diff
		// weight_incs[n_hidden x n_visible] = h_buf[n_hidden x n_samples] x h_diff[n_samples x n_visible]
		calc_weight_incs_gpu<<<dim_grid, dim_threads>>>(n_hidden, n_visible, n_samples, d_diff, d_buf, d_weight_incs);
		hipDeviceSynchronize();

		d_x = (n_hidden % threads_width) ? (n_hidden/threads_width+1) : (n_hidden/threads_width);
		d_y = (n_samples % threads_width) ? (n_samples/threads_width+1) : (n_samples/threads_width);
		dim_threads = dim3(threads_width, threads_width);
		dim_grid = dim3(d_x, d_y);

		// bias_incs[n_hidden] += h_diff[n_hidden]
		// Sum reduce the value of each hidden layer's node for all samples
		weight_incs_add_diff_gpu<<<dim_grid, dim_threads>>>(n_samples, n_hidden, d_diff, d_weight_incs + n_visible*n_hidden);
		hipDeviceSynchronize();

	}

	// copy weight_incs result from gpu memory to host memory
	hipMemcpy(&weight_incs[0], d_weight_incs_g, weight_incs.size() * sizeof(float), hipMemcpyDeviceToHost);

	delete[] h_buf;
	delete[] h_diff;

	hipFree(d_buf_g);
	hipFree(d_weight_g);
	hipFree(d_hidden_bias_g);
	hipFree(d_diff_g);
	hipFree(d_weight_incs_g);

	auto cend = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(cend - cstart).count() / 1000.0;
	std::cout << "evaluating: cost=" << cost << ", error=" << error / n_samples << " in " << duration << "ms" << std::endl;

	endTimer("##### gradient()-calc gradient");
	endTimer("#### gradient()");
	return 0;
}

int DeepBeliefNet::fine_tune(std::vector<Vector>& inputs, std::vector<Vector>& targets, 
		LRBM::Conf& conf, bool is_cuda /* = false */) 
{
	int batch_size = conf.batch_size_, max_epoch = conf.max_epoch_, max_batches = conf.max_batches_; 
	int max_layer = this->rbms_.size();

	startTimer("## fine_tune()");
	std::vector<std::vector<Vector>> probs(max_layer);
	for (int i = 0; i < max_layer; ++i) {
		const RBMP& rbm = this->rbms_[i];
		probs[i].resize(batch_size);
		for (auto &v: probs[i]) { v.resize(rbm->num_hidden()); }
	}

	// max_epoch indicates how many times to repeat the learning of max_batches
	// max_batches indicates min(max_batches_, total_input/batch_size)
	// batch_size indicates number of input data's used for mini-batch learning (SGD)
	for (int epoch = 0; epoch < max_epoch; ++epoch) {
		for (int j = 0; j < max_batches; ++j) {
			int start = j * batch_size, end = start + std::min(batch_size, int(inputs.size()) - j * batch_size);
			std::cout << "epoch: " << epoch << ", batch: " << j << ", samples: "<< (end - start) << std::endl;
			GradientContext ctx(Batch(inputs.begin() + start, inputs.begin() + end), probs, epoch);
			//        ctx.start_layer_ = (epoch > std::min(6, max_epoch / 2)? 0: this->rbms_.size() - 1);
			if (! targets.empty())
				ctx.targets_ = Batch(targets.begin() + start, targets.begin() + end);
			this->minimize(ctx, is_cuda);
		}
	}

	endTimer("## fine_tune()");
	return 0;
}

	template <class Vector1, class Vector2, class Vector3>
const Vector2& RBM::activate_hidden_gpu(const Vector1& visible, Vector2& hidden, const Vector3& bias_hidden, const Vector3& weight, Type type)
{
	startTimer("##### activate_hidden()");
	size_t n_visible = visible.size(), n_hidden = hidden.size();
	std::fill(hidden.begin(), hidden.end(), 0);

	float *d_visible, *d_weight, *d_hidden, *d_bias;
	hipMalloc((void **)&d_visible, n_visible * sizeof(float));
	hipMalloc((void **)&d_weight, n_hidden * n_visible * sizeof(float));
	hipMalloc((void **)&d_bias, n_hidden * sizeof(float));
	hipMalloc((void **)&d_hidden, n_hidden * sizeof(float));
	hipMemcpy(d_visible, &visible[0], n_visible * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, &weight[0], n_hidden * n_visible * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_bias, &bias_hidden[0], n_hidden * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_hidden, 0, n_hidden * sizeof(float));

	int threads_width=512;
	int d_x = (n_hidden % threads_width) ? (n_hidden/threads_width+1) : (n_hidden/threads_width);
	int d_y = 1;
	dim3 dim_threads(threads_width, 1);
	dim3 dim_grid(d_x, d_y);

	if (type == Type::SIGMOID)
		dot_gpu<<<dim_grid, dim_threads>>>(n_hidden, n_visible, d_weight, d_visible, d_hidden, d_bias, 1);
	else if (type == Type::EXP)
		dot_gpu<<<dim_grid, dim_threads>>>(n_hidden, n_visible, d_weight, d_visible, d_hidden, d_bias, 0);
	hipDeviceSynchronize();
	hipMemcpy(&hidden[0], d_hidden, n_hidden * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_visible);
	hipFree(d_weight);
	hipFree(d_hidden);
	hipFree(d_bias);

	endTimer("##### activate_hidden()");
	return hidden;
}
