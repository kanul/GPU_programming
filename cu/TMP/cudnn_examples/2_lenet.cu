#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <opencv2/opencv.hpp>

#define getMillisecond(start, end) \
    (end.tv_sec-start.tv_sec)*1000 + \
    (end.tv_usec-start.tv_usec)/1000.0

#define checkCUDA(expression)                             \
{                                                         \
	hipError_t status = (expression);                      \
	if (status != hipSuccess) {                            \
		printf("Error on line %d: err code %d (%s)\n",        \
				__LINE__, status, hipGetErrorString(status));    \
		exit(EXIT_FAILURE);                                   \
	}                                                       \
}

#define checkCUBLAS(expression)                           \
{                                                         \
	hipblasStatus_t status = (expression);                   \
	if (status != HIPBLAS_STATUS_SUCCESS) {                  \
		printf("Error on line %d: err code %d\n",             \
				__LINE__, status);                                \
		exit(EXIT_FAILURE);                                   \
	}                                                       \
}

#define checkCUDNN(expression)                            \
{                                                         \
	hipdnnStatus_t status = (expression);                    \
	if (status != HIPDNN_STATUS_SUCCESS) {                   \
		printf("Error on line %d: err code %d (%s)\n",        \
				__LINE__, status, hipdnnGetErrorString(status));   \
		exit(EXIT_FAILURE);                                   \
	}                                                       \
}

cv::Mat load_image(const char* image_path) {
	cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_GRAYSCALE);
	image.convertTo(image, CV_32FC1);
	cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
	return image;
}

bool load_weight(float* p_weight, int elemCount, const char* filename)
{
	// Read weights file
	FILE *fp = fopen(filename, "rb");
	if (!fp)
	{
		printf("ERROR: Cannot open file %s\n", filename);
		return false;
	}
	fread(p_weight, sizeof(float), elemCount, fp);
	fclose(fp);

	return true;
}


int main(int argc, char const *argv[]) {
	hipblasHandle_t cublas;
	hipdnnHandle_t cudnn;
	checkCUBLAS(hipblasCreate(&cublas));
	checkCUDNN(hipdnnCreate(&cudnn));

	int    batch_size = 1;
	size_t workspace_bytes = 0;
	cv::Mat image = load_image("image/input.pgm");

	/* Input */

	int input_dim = 28;
	int input_channels = 1;

	float* d_input{NULL};
	int input_bytes = batch_size * input_channels * input_dim * input_dim
										* sizeof(float);
	hipMalloc(&d_input, input_bytes);
	hipMemcpy(d_input, image.ptr<float>(0), input_bytes, hipMemcpyHostToDevice);

	// Input Tensor
	hipdnnTensorDescriptor_t input_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/batch_size,
				/*channels=*/input_channels,
				/*height=*/input_dim,
				/*width=*/input_dim));

	/* Layer 1. Convolution */

	int l1_kernel_dim = 5;
	int l1_pad = 0;
	int l1_stride = 1;
	int l1_dilation = 1;

	int l1_out_dim = 24;
	int l1_out_channels = 20;

	char* l1_weight_file = "pretrained/conv1.bin";
	char* l1_weight_bias_file = "pretrained/conv1.bias.bin";

	// Describing Operands

	hipdnnTensorDescriptor_t l1_out_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l1_out_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l1_out_descriptor,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/batch_size,
				/*channels=*/l1_out_channels,
				/*height=*/l1_out_dim,
				/*width=*/l1_out_dim));

	hipdnnTensorDescriptor_t l1_kernel_bias_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l1_kernel_bias_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l1_kernel_bias_descriptor,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/1,
				/*channels=*/l1_out_channels,
				/*height=*/1,
				/*width=*/1));

	hipdnnFilterDescriptor_t l1_kernel_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&l1_kernel_descriptor));
	checkCUDNN(hipdnnSetFilter4dDescriptor(l1_kernel_descriptor,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*out_channels=*/l1_out_channels,
				/*in_channels=*/input_channels,
				/*kernel_height=*/l1_kernel_dim,
				/*kernel_width=*/l1_kernel_dim));

	// Describing the Convolution Kernel

	hipdnnConvolutionDescriptor_t l1_convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&l1_convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(l1_convolution_descriptor,
				/*pad_height=*/l1_pad,
				/*pad_width=*/l1_pad,
				/*vertical_stride=*/l1_stride,
				/*horizontal_stride=*/l1_stride,
				/*dilation_height=*/l1_dilation,
				/*dilation_width=*/l1_dilation,
				/*mode=*/HIPDNN_CROSS_CORRELATION,
				/*dataType=*/HIPDNN_DATA_FLOAT
				));

	hipdnnConvolutionFwdAlgo_t l1_convolution_algorithm;
	checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(cudnn,
				input_descriptor,
				l1_kernel_descriptor,
				l1_convolution_descriptor,
				l1_out_descriptor,
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
				/*memoryLimitInBytes=*/0,
				&l1_convolution_algorithm));

	size_t l1_workspace_bytes = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
				input_descriptor,
				l1_kernel_descriptor,
				l1_convolution_descriptor,
				l1_out_descriptor,
				l1_convolution_algorithm,
				&l1_workspace_bytes));
	workspace_bytes = max(workspace_bytes, l1_workspace_bytes);

	/* Allocating Memory for Layer 1 */

	int l1_out_bytes = batch_size * l1_out_channels * l1_out_dim * l1_out_dim
										 * sizeof(float);
	float* d_l1_output{NULL};
	hipMalloc(&d_l1_output, l1_out_bytes);
	hipMemset(d_l1_output, 0, l1_out_bytes);

	int l1_kernel_bytes = input_channels * l1_out_channels * l1_kernel_dim
												* l1_kernel_dim * sizeof(float);
	float* l1_kernel = (float*)malloc (l1_kernel_bytes);
	float* l1_kernel_bias = (float*)malloc (l1_out_channels * sizeof(float));

	// load pretrained weight
	load_weight(l1_kernel,
							input_channels * l1_out_channels * l1_kernel_dim * l1_kernel_dim,
							l1_weight_file);
	load_weight(l1_kernel_bias, l1_out_channels, l1_weight_bias_file);

	float* d_l1_kernel{NULL};
	hipMalloc(&d_l1_kernel, l1_kernel_bytes);
	hipMemcpy(d_l1_kernel, l1_kernel, l1_kernel_bytes, hipMemcpyHostToDevice);

	float* d_l1_kernel_bias{NULL};
	hipMalloc(&d_l1_kernel_bias, l1_out_channels * sizeof(float));
	hipMemcpy(d_l1_kernel_bias, l1_kernel_bias, l1_out_channels * sizeof(float),
						 hipMemcpyHostToDevice);



	/* Layer 2. Max Pooling */

	int l2_pool_dim = 2;
	int l2_pad = 0;
	int l2_stride = 2;

	int l2_out_dim = (l1_out_dim + l2_pad*2) / l2_stride;
	int l2_out_channels = l1_out_channels;

	// Describing Operands
	hipdnnTensorDescriptor_t l2_out_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l2_out_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l2_out_descriptor,
				/*format=*/HIPDNN_TENSOR_NCHW,
				/*dataType=*/HIPDNN_DATA_FLOAT,
				/*batch_size=*/batch_size,
				/*channels=*/l2_out_channels,
				/*height=*/l2_out_dim,
				/*width=*/l2_out_dim));

	hipdnnPoolingDescriptor_t l2_pool_descriptor;
	checkCUDNN(hipdnnCreatePoolingDescriptor(&l2_pool_descriptor));
	checkCUDNN(hipdnnSetPooling2dDescriptor(l2_pool_descriptor,
				/*poolingMode=*/HIPDNN_POOLING_MAX,
				/*NanPropagationMode=*/HIPDNN_PROPAGATE_NAN,
				l2_pool_dim, l2_pool_dim,
				l2_pad,      l2_pad,
				l2_stride,   l2_stride));

	/* Allocating Memory for Layer 2 */

	int l2_out_bytes = batch_size * l2_out_channels * l2_out_dim * l2_out_dim
										 * sizeof(float);

	float* d_l2_output{NULL};
	hipMalloc(&d_l2_output, l2_out_bytes);
	hipMemset(d_l2_output, 0, l2_out_bytes);



	/* Layer 3. Convolution */
	int l3_kernel_dim =  
	int l3_pad = 
	int l3_stride = 
	int l3_dilation = 

	int l3_out_dim = 
	int l3_out_channels = 

	char* l3_weight_file = "pretrained/conv2.bin";
	char* l3_weight_bias_file = "pretrained/conv2.bias.bin";

	// Describing Operands

	hipdnnTensorDescriptor_t l3_out_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l3_out_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l3_out_descriptor,
				/*format=*/  ,
				/*dataType=*/  ,
				/*batch_size=*/  ,
				/*channels=*/  ,
				/*height=*/  ,
				/*width=*/  ));

	hipdnnTensorDescriptor_t l3_kernel_bias_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l3_kernel_bias_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l3_kernel_bias_descriptor,
				/*format=*/  ,
				/*dataType=*/  ,
				/*batch_size=*/  ,
				/*channels=*/  ,
				/*height=*/  ,
				/*width=*/  ));

	hipdnnFilterDescriptor_t l3_kernel_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&l3_kernel_descriptor));
	checkCUDNN(hipdnnSetFilter4dDescriptor(l3_kernel_descriptor,
				/*dataType=*/  ,
				/*format=*/  ,
				/*out_channels=*/  ,
				/*in_channels=*/  ,
				/*kernel_height=*/  ,
				/*kernel_width=*/  ));

	// Describing the Convolution Kernel

	hipdnnConvolutionDescriptor_t l3_convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&l3_convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(l3_convolution_descriptor,
				/*pad_height=*/  ,
				/*pad_width=*/  ,
				/*vertical_stride=*/  ,
				/*horizontal_stride=*/  ,
				/*dilation_height=*/  ,
				/*dilation_width=*/  ,
				/*mode=*/  ,
				/*dataType=*/  ));

	hipdnnConvolutionFwdAlgo_t l3_convolution_algorithm;
	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
				,
				,
				,
				,
				,
				/*memoryLimitInBytes=*/0,
				));

	size_t l3_workspace_bytes = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
				,
				,
				,
				,
				,
				));
	workspace_bytes = max(workspace_bytes, l3_workspace_bytes);

	/* Allocating Memory for Layer 3 */

	int l3_out_bytes = 
	float* d_l3_output{NULL};
	hipMalloc(&d_l3_output, l3_out_bytes);
	hipMemset(d_l3_output, 0, l3_out_bytes);

	int l3_kernel_bytes = 
	float* l3_kernel = (float*)malloc (l3_kernel_bytes);
	float* l3_kernel_bias = (float*)malloc ( );

	// load pretrained weight
	load_weight(l3_kernel,
							,
							l3_weight_file);
	load_weight(l3_kernel_bias,   , l3_weight_bias_file);

	float* d_l3_kernel{NULL};
	hipMalloc(&d_l3_kernel, l3_kernel_bytes);
	hipMemcpy(d_l3_kernel, l3_kernel, l3_kernel_bytes, hipMemcpyHostToDevice);

	float* d_l3_kernel_bias{NULL};
	hipMalloc(&d_l3_kernel_bias, );
	hipMemcpy(d_l3_kernel_bias,
						 l3_kernel_bias,
						 ,
						 hipMemcpyHostToDevice);



	/* Layer 4. Max Pooling */

	int l4_pool_dim = 
	int l4_pad = 
	int l4_stride = 

	int l4_out_dim = 
	int l4_out_channels = 

	// Describing Operands
	hipdnnTensorDescriptor_t l4_out_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l4_out_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l4_out_descriptor,
				/*format=*/  ,
				/*dataType=*/  ,
				/*batch_size=*/  ,
				/*channels=*/  ,
				/*height=*/  ,
				/*width=*/  ));

	hipdnnPoolingDescriptor_t l4_pool_descriptor;
	checkCUDNN(hipdnnCreatePoolingDescriptor(&l4_pool_descriptor));
	checkCUDNN(hipdnnSetPooling2dDescriptor(l4_pool_descriptor,
				/*poolingMode=*/HIPDNN_POOLING_MAX,
				/*NanPropagationMode=*/HIPDNN_PROPAGATE_NAN,
				, ,
				, ,
				, ));

	/* Allocating Memory for Layer 2 */

	int l4_out_bytes = 

	float* d_l4_output{NULL};
	hipMalloc(&d_l4_output, l4_out_bytes);
	hipMemset(d_l4_output, 0, l4_out_bytes);



	/* Layer 5. Fully Connected Layer */
	int l5_fc_in_dim = (l4_out_channels * l4_out_dim * l4_out_dim);
	int l5_fc_out_dim = 500;
	int l5_fc_neuron_size = l5_fc_in_dim * l5_fc_out_dim;

	char* l5_weight_file = "pretrained/ip1.bin";
	char* l5_weight_bias_file = "pretrained/ip1.bias.bin";

	hipdnnActivationDescriptor_t l5_fc_activation_descriptor;
	checkCUDNN(hipdnnCreateActivationDescriptor(&l5_fc_activation_descriptor));
	checkCUDNN(hipdnnSetActivationDescriptor(l5_fc_activation_descriptor,
																					HIPDNN_ACTIVATION_RELU,
																					HIPDNN_PROPAGATE_NAN, 0.0));

	hipdnnTensorDescriptor_t l5_relu_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l5_relu_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l5_relu_descriptor,
					HIPDNN_TENSOR_NCHW,
					HIPDNN_DATA_FLOAT,
					batch_size, l5_fc_out_dim, 1, 1));

	/* Allocating memory for Layer 5 */

	float* l5_fc_neuron = (float*)malloc (l5_fc_neuron_size * sizeof(float));
	load_weight(l5_fc_neuron, l5_fc_neuron_size, l5_weight_file);

	float* l5_fc_neuron_bias = (float*)malloc (l5_fc_out_dim * sizeof(float));
	load_weight(l5_fc_neuron_bias, l5_fc_out_dim, l5_weight_bias_file);

	float* d_l5_fc_neuron{NULL};
	checkCUDA(hipMalloc(&d_l5_fc_neuron, l5_fc_neuron_size * sizeof(float)));
	checkCUDA(hipMemcpy(d_l5_fc_neuron, l5_fc_neuron,
			l5_fc_neuron_size * sizeof(float), hipMemcpyHostToDevice));

	float* d_l5_fc_neuron_bias{NULL};
	checkCUDA(hipMalloc(&d_l5_fc_neuron_bias, l5_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemcpy(d_l5_fc_neuron_bias, l5_fc_neuron_bias,
			l5_fc_out_dim * sizeof(float), hipMemcpyHostToDevice));

	float* d_l5_fc_output{NULL};
	checkCUDA(hipMalloc(&d_l5_fc_output,
											 batch_size * l5_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemset(d_l5_fc_output, 0, l5_fc_out_dim * sizeof(float)));

	float* d_l5_relu_output{NULL};
	checkCUDA(hipMalloc(&d_l5_relu_output,
											 batch_size * l5_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemset(d_l5_relu_output, 0, l5_fc_out_dim * sizeof(float)));



	/* Layer 6. Fully Connected Layer */
	int l6_fc_in_dim = l5_fc_out_dim;
	int l6_fc_out_dim = 10;
	int l6_fc_neuron_size = l6_fc_in_dim * l6_fc_out_dim;

	char* l6_weight_file = "pretrained/ip2.bin";
	char* l6_weight_bias_file = "pretrained/ip2.bias.bin";

	hipdnnActivationDescriptor_t l6_fc_activation_descriptor;
	checkCUDNN(hipdnnCreateActivationDescriptor(&l6_fc_activation_descriptor));
	checkCUDNN(hipdnnSetActivationDescriptor(l6_fc_activation_descriptor,
																					HIPDNN_ACTIVATION_RELU,
																					HIPDNN_PROPAGATE_NAN, 0.0));

	hipdnnTensorDescriptor_t l6_softmax_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&l6_softmax_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(l6_softmax_descriptor,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				batch_size, l6_fc_out_dim, 1, 1));

	/* Allocating memory for Layer 6 */

	float* l6_fc_neuron = (float*)malloc (l6_fc_neuron_size * sizeof(float));
	load_weight(l6_fc_neuron, l6_fc_neuron_size, l6_weight_file);

	float* l6_fc_neuron_bias = (float*)malloc (l6_fc_out_dim * sizeof(float));
	load_weight(l6_fc_neuron_bias, l6_fc_out_dim, l6_weight_bias_file);

	float* l6_softmax_output = (float*)malloc (l6_fc_out_dim * sizeof(float));

	float* d_l6_fc_neuron{NULL};
	checkCUDA(hipMalloc(&d_l6_fc_neuron, l6_fc_neuron_size * sizeof(float)));
	checkCUDA(hipMemcpy(d_l6_fc_neuron, l6_fc_neuron,
			l6_fc_neuron_size * sizeof(float), hipMemcpyHostToDevice));

	float* d_l6_fc_neuron_bias{NULL};
	checkCUDA(hipMalloc(&d_l6_fc_neuron_bias, l6_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemcpy(d_l6_fc_neuron_bias, l6_fc_neuron_bias,
			l6_fc_out_dim * sizeof(float), hipMemcpyHostToDevice));

	float* d_l6_fc_output{NULL};
	checkCUDA(hipMalloc(&d_l6_fc_output,
											 batch_size * l6_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemset(d_l6_fc_output, 0, l6_fc_out_dim * sizeof(float)));

	float* d_l6_softmax_output{NULL};
	checkCUDA(hipMalloc(&d_l6_softmax_output,
											 batch_size * l6_fc_out_dim * sizeof(float)));
	checkCUDA(hipMemset(d_l6_softmax_output, 0, l6_fc_out_dim * sizeof(float)));



	/* Forward */

	struct timeval start, end;
	gettimeofday(&start, NULL);

	// Allocating Memory for Workspace
	void* d_workspace{NULL};
	hipMalloc(&d_workspace, workspace_bytes);

	// One vector for FC
	float *d_onevec{NULL};
	checkCUDA(hipMalloc(&d_onevec, batch_size * sizeof(float)));
	checkCUDA(hipMemset(d_onevec, 1, batch_size * sizeof(float)));

	/* Layer 1. Convolution */
	const float alpha = 1, beta = 0;
	checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor,
				                             /*input device mem=*/d_input,
				                             l1_kernel_descriptor,
				                             /*kernel device mem*/d_l1_kernel,
				                             l1_convolution_descriptor,
				                             l1_convolution_algorithm,
				                             d_workspace, workspace_bytes,
				                             &beta, l1_out_descriptor,
				                             /*output device mem=*/d_l1_output));
	// Add bias
	checkCUDNN(hipdnnAddTensor(cudnn, &alpha, l1_kernel_bias_descriptor,
                            d_l1_kernel_bias, &alpha, l1_out_descriptor,
                            d_l1_output));

	/* Layer 2. Max Pooling */
	checkCUDNN(hipdnnPoolingForward(cudnn, l2_pool_descriptor, &alpha,
																 l1_out_descriptor, d_l1_output, &beta,
																 l2_out_descriptor, d_l2_output));

	/* Layer 3. Convolution */
	checkCUDNN(hipdnnConvolutionForward(cudnn,
				                             &alpha,
				                             l2_out_descriptor,
				                             /*input device mem=*/d_l2_output,
				                             l3_kernel_descriptor,
				                             /*kernel device mem*/d_l3_kernel,
				                             l3_convolution_descriptor,
				                             l3_convolution_algorithm,
				                             d_workspace,
				                             workspace_bytes,
				                             &beta,
				                             l3_out_descriptor,
				                             /*output device mem=*/d_l3_output));
	// Add bias
	checkCUDNN(hipdnnAddTensor(cudnn, &alpha, l3_kernel_bias_descriptor,
                            d_l3_kernel_bias, &alpha, l3_out_descriptor,
                            d_l3_output));

	/* Layer 4. Max Pooling */
	checkCUDNN(hipdnnPoolingForward(cudnn, l4_pool_descriptor, &alpha,
																 l3_out_descriptor, d_l3_output, &beta,
																 l4_out_descriptor, d_l4_output));

	/* Layer 5. Fully Connected */
	// FC1 layer
	// Forward propagate neurons using weights
	checkCUBLAS(hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
				l5_fc_out_dim, batch_size, l5_fc_in_dim,
				&alpha,
				d_l5_fc_neuron, l5_fc_in_dim,
				d_l4_output, l5_fc_in_dim,
				&beta,
				d_l5_fc_output, l5_fc_out_dim));
	// Add bias using GEMM's "beta"
	checkCUBLAS(hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
				l5_fc_out_dim, batch_size, 1,
				&alpha,
				d_l5_fc_neuron_bias, l5_fc_out_dim,
				d_onevec, 1,
				&alpha,
				d_l5_fc_output, l5_fc_out_dim));
	// ReLU activation
	checkCUDNN(hipdnnActivationForward(cudnn, l5_fc_activation_descriptor, &alpha,
																		l5_relu_descriptor, d_l5_fc_output, &beta,
																		l5_relu_descriptor, d_l5_relu_output));

	/* Layer 6. Fully Connected (Softmax) */
	// FC2 layer
	checkCUBLAS(hipblasSgemm(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
				l6_fc_out_dim, batch_size, l6_fc_in_dim,
				&alpha,
				d_l6_fc_neuron, l6_fc_in_dim,
				d_l5_relu_output, l6_fc_in_dim,
				&beta,
				d_l6_fc_output, l6_fc_out_dim));
	// Add bias using GEMM's "beta"
	checkCUBLAS(hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
				l6_fc_out_dim, batch_size, 1,
				&alpha,
				d_l6_fc_neuron_bias, l6_fc_out_dim,
				d_onevec, 1,
				&alpha,
				d_l6_fc_output, l6_fc_out_dim));
	// Softmax loss
	checkCUDNN(hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE,
																 HIPDNN_SOFTMAX_MODE_CHANNEL,
																 &alpha, l6_softmax_descriptor,
																 d_l6_fc_output, &beta, l6_softmax_descriptor,
																 d_l6_softmax_output));

	/* Show result */

	checkCUDA(hipMemcpy(l6_softmax_output, d_l6_softmax_output,
											 l6_fc_out_dim * sizeof(float), hipMemcpyDeviceToHost));

	gettimeofday(&end, NULL);

	int i, chosen = 0;
	for (i = 0; i < l6_fc_out_dim; i++) {
		printf("%d: %.2f\n", i, l6_softmax_output[i]);
		if (l6_softmax_output[i] > l6_softmax_output[chosen])
			chosen = i;
	}
	printf("\nPredict: %d\n", chosen);
	printf("Time: %f\n", getMillisecond(start, end));



	/* Free */

	// input
	hipdnnDestroyTensorDescriptor(input_descriptor);
	hipFree(d_input);

	// Layer 1
	hipdnnDestroyTensorDescriptor(l1_out_descriptor);
	hipdnnDestroyFilterDescriptor(l1_kernel_descriptor);
	hipdnnDestroyTensorDescriptor(l1_kernel_bias_descriptor);
	hipdnnDestroyConvolutionDescriptor(l1_convolution_descriptor);
	hipFree(d_l1_output);
	hipFree(d_l1_kernel);
	hipFree(d_l1_kernel_bias);
	free(l1_kernel);
	free(l1_kernel_bias);

	// Layer 2
	hipdnnDestroyTensorDescriptor(l2_out_descriptor);
	hipdnnDestroyPoolingDescriptor(l2_pool_descriptor);
	hipFree(d_l2_output);

	// Layer 3
	hipdnnDestroyTensorDescriptor(l3_out_descriptor);
	hipdnnDestroyFilterDescriptor(l3_kernel_descriptor);
	hipdnnDestroyTensorDescriptor(l3_kernel_bias_descriptor);
	hipdnnDestroyConvolutionDescriptor(l3_convolution_descriptor);
	hipFree(d_l3_output);
	hipFree(d_l3_kernel);
	hipFree(d_l3_kernel_bias);
	free(l3_kernel);
	free(l3_kernel_bias);

	// Layer 4
	hipdnnDestroyTensorDescriptor(l4_out_descriptor);
	hipdnnDestroyPoolingDescriptor(l4_pool_descriptor);
	hipFree(d_l4_output);

	// Layer 5
	checkCUDNN(hipdnnDestroyActivationDescriptor(l5_fc_activation_descriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(l5_relu_descriptor));
	checkCUDA(hipFree(d_l5_fc_output));
	checkCUDA(hipFree(d_l5_fc_neuron));
	checkCUDA(hipFree(d_l5_fc_neuron_bias));
	checkCUDA(hipFree(d_l5_relu_output));
	free(l5_fc_neuron);
	free(l5_fc_neuron_bias);

	// Layer 6
	checkCUDNN(hipdnnDestroyActivationDescriptor(l6_fc_activation_descriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(l6_softmax_descriptor));
	checkCUDA(hipFree(d_l6_fc_output));
	checkCUDA(hipFree(d_l6_fc_neuron));
	checkCUDA(hipFree(d_l6_fc_neuron_bias));
	checkCUDA(hipFree(d_l6_softmax_output));
	free(l6_fc_neuron);
	free(l6_fc_neuron_bias);
	free(l6_softmax_output);

	// etc
	hipFree(d_onevec);
	hipFree(d_workspace);
	hipblasDestroy(cublas);
	hipdnnDestroy(cudnn);

	return 0;
}
