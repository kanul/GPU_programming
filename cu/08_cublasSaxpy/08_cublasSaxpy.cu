
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#define N 1000000

#define checkCUDA(expression)						\
{													\
	hipError_t status = (expression);				\
	if (status != hipSuccess) {					\
		printf("Error on line %d: err code %d\n",	\
				__LINE__, status);					\
		exit(EXIT_FAILURE);							\
	}												\
}

#define checkCUBLAS(expression)						\
{													\
	hipblasStatus_t status = (expression);			\
	if (status != HIPBLAS_STATUS_SUCCESS) {			\
		printf("Error on line %d: err code %d\n",	\
				__LINE__, status);					\
		exit(EXIT_FAILURE);							\
	}												\
}

#define getMillisecond(start, end) (end.tv_sec-start.tv_sec) * 1000 + (end.tv_usec-start.tv_usec) / 1000.0

int main(void) {
	hipblasHandle_t handle;
	float *x, *y;
	float *devPtrX, *devPtrY;
	float alpha = 1.2;
	float incx = 1, incy = 1;

	float ms = 0;
	struct timeval start, end;
	srand(2018);

	x = (float *)malloc (N*sizeof(float));
	y = (float *)malloc (N*sizeof(float));

	float *resultCPU = (float *)malloc(N*sizeof(float));
	float *resultGPU = (float *)malloc(N*sizeof(float));
	
	for (int i = 0; i < N; i++) {
		x[i] = (rand() % 1000000) / 10000.0;
		y[i] = (rand() % 1000000) / 10000.0;
	}

	checkCUDA(hipMalloc((void**)&devPtrX, N * sizeof(float)));
	checkCUDA(hipMalloc((void**)&devPtrY, N * sizeof(float)));

	checkCUBLAS(hipblasCreate(&handle));

	checkCUBLAS(hipblasSetVector (N, sizeof(float), x, 1, devPtrX, 1));
	checkCUBLAS(hipblasSetVector (N, sizeof(float), y, 1, devPtrY, 1));

	gettimeofday(&start, NULL);
	checkCUBLAS(hipblasSaxpy (handle, N, &alpha, devPtrX, incx, devPtrY, incy));
	gettimeofday(&end, NULL);

	ms = getMillisecond(start, end);
	printf("GPU time: %f (ms)\n", ms);

	checkCUBLAS (hipblasGetVector(N, sizeof(float), devPtrY, 1, resultGPU, 1));

	gettimeofday(&start, NULL);
	for(int i = 0; i < N; i++) {
		resultCPU[i] = x[i]*alpha + y[i];
	}

	gettimeofday(&end, NULL);

	ms = getMillisecond(start, end);
	printf("CPU time: %f (ms)\n", ms);

	float error = 0;
	for (int i = 0 ; i < N; i++) {
		error += abs((resultCPU[i] - resultGPU[i]) / resultCPU[i]);
	}

	error = error / N * 100;
	printf("Mean Absolute Percentage Error: %f (%%)\n", error);

	checkCUDA(hipFree(devPtrX));
	checkCUDA(hipFree(devPtrY));
	checkCUBLAS(hipblasDestroy(handle));
	free(x);
	free(y);
	free(resultCPU);
	free(resultGPU);
	return EXIT_SUCCESS;
}


